#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
extern "C" {
__global__ void prepare(hiprandState *state, unsigned int *global_ta_state, int *clause_weights) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState localState = state[index];

    for (int clause = index; clause < CLAUSES; clause += stride) {
        unsigned int *ta_state = &global_ta_state[clause * LA_CHUNKS * STATE_BITS];

        for (int la_chunk = 0; la_chunk < LA_CHUNKS; ++la_chunk) {
            for (int b = 0; b < STATE_BITS - 1; ++b) {
                ta_state[la_chunk * STATE_BITS + b] = ~0;
            }
            ta_state[la_chunk * STATE_BITS + STATE_BITS - 1] = 0;
        }
    }

    for (int clause = 0; clause < CLAUSES; clause++) {
        for (int class_id = 0; class_id < CLASSES; ++class_id) {
            if (NEGATIVE_CLAUSES)
                clause_weights[class_id * CLAUSES + clause] = 1 - 2 * (hiprand(&localState) % 2);
            else
                clause_weights[class_id * CLAUSES + clause] = 1;
        }
    }

    state[index] = localState;
}

__global__ void prepare_packed(hiprandState *state, unsigned int *global_ta_state, unsigned int *included_literals,
                               unsigned int *included_literals_length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState localState = state[index];

    for (int clause = index; clause < CLAUSES; clause += stride) {
        unsigned int *ta_state = &global_ta_state[clause * LA_CHUNKS * STATE_BITS];

        included_literals_length[clause] = 0;
        for (int literal = 0; literal < FEATURES; ++literal) {
            int chunk = literal / INT_SIZE;
            int pos = literal % INT_SIZE;

            if ((ta_state[chunk * STATE_BITS + STATE_BITS - 1] & (1U << pos)) > 0) {
                included_literals[clause * FEATURES * 2 + included_literals_length[clause] * 2] = literal;
                included_literals_length[clause]++;
            }
        }
    }
    state[index] = localState;
}
}
