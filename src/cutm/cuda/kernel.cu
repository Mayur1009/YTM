#include "hip/hip_runtime.h"
// Ignore this block, it is used to only for neovim clangd lsp.
#ifdef IS_NEOVIM_CLANGD_ENV
#define CLAUSES 100
#define THRESH 500
#define S 10
#define Q 1
#define DIM0 28
#define DIM1 28
#define DIM2 1
#define PATCH_DIM0 10
#define PATCH_DIM1 10
#define PATCHES 361
#define LITERALS 272
#define MAX_INCLUDED_LITERALS 272
#define APPEND_NEGATED 1
#define INIT_NEG_WEIGHTS 1
#define NEGATIVE_CLAUSES 1
#define CLASSES 10
#define MAX_TA_STATE 255
#define ENCODE_LOC 1
#endif

#include <hiprand/hiprand_kernel.h>

#include <cstdint>

#define VECTORIZED_LIMIT (LITERALS & ~3)
#define S_INV (1.0f / S)
#define Q_PROB (1.0f * Q / max(1, CLASSES - 1))
#define HALF_STATE (MAX_TA_STATE / 2)
#define INT_SIZE 32
#define NUM_LITERAL_CHUNKS (((LITERALS - 1) / INT_SIZE) + 1)
#if ((LITERALS % INT_SIZE) != 0)
#define FILTER (~(0xFFFFFFFF << (LITERALS % INT_SIZE)))
#else
#define FILTER 0xFFFFFFFF
#endif

typedef unsigned long long ull;

extern "C" {
__global__ void initialize(hiprandState *rng, unsigned int *global_ta_states, float *clause_weights) {
    /*
     * Initialize the TA states to middle state and clause weights(randomly to -1 and 1).
     */
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState localState = rng[index];

    for (int clause = index; clause < CLAUSES; clause += stride) {
        for (int li = 0; li < LITERALS; ++li) {
            global_ta_states[clause * LITERALS + li] = HALF_STATE;  // Initialize TA states to 0
        }
        for (int class_id = 0; class_id < CLASSES; ++class_id) {
#if INIT_NEG_WEIGHTS
            clause_weights[clause * CLASSES + class_id] = (1.0f - 2.0f * (float)(hiprand(&localState) % 2));
#else
            clause_weights[clause * CLASSES + class_id] = 1.0f;
#endif
        }
    }
    rng[index] = localState;
}

__global__ void encode_batch(const unsigned int *X, unsigned int *encoded_X, const int N) {
    // X -> (N * DIM0 * DIM1 * DIM2)
    // encoded_X -> (N * PATCHES * NUM_LITERAL_CHUNKS)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (ull e_patch = index; e_patch < (ull)(PATCHES * N); e_patch += stride) {
        ull e = e_patch / PATCHES;
        ull patch_id = e_patch % PATCHES;

        // Calculate the starting point of the patch in the original image
        int patch_coordinate_y = patch_id / (DIM0 - PATCH_DIM0 + 1);
        int patch_coordinate_x = patch_id % (DIM0 - PATCH_DIM0 + 1);

        ull encX_offset = e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * (ull)NUM_LITERAL_CHUNKS;
        unsigned int *patch_output = &encoded_X[encX_offset];

#if APPEND_NEGATED
#pragma unroll 4
        for (int literal = LITERALS / 2; literal < LITERALS; ++literal) {
            int chunk_nr = literal / INT_SIZE;
            int chunk_pos = literal % INT_SIZE;
            patch_output[chunk_nr] |= (1u << chunk_pos);
        }
#endif

#pragma unroll 4
        for (int lit = 0; lit < patch_coordinate_y; ++lit) {
            int chunk_nr = lit / INT_SIZE;
            int chunk_pos = lit % INT_SIZE;
            patch_output[chunk_nr] |= (1u << chunk_pos);
#if APPEND_NEGATED
            int neg_chunk_nr = (lit + (LITERALS / 2)) / INT_SIZE;
            int neg_chunk_pos = (lit + (LITERALS / 2)) % INT_SIZE;
            patch_output[neg_chunk_nr] &= ~(1u << neg_chunk_pos);
#endif
        }

#pragma unroll 4
        for (int lit = 0; lit < patch_coordinate_x; ++lit) {
            int chunk_nr = (DIM1 - PATCH_DIM1 + lit) / INT_SIZE;
            int chunk_pos = (DIM1 - PATCH_DIM1 + lit) % INT_SIZE;
            patch_output[chunk_nr] |= (1u << chunk_pos);
#if APPEND_NEGATED
            int neg_chunk_nr = ((DIM1 - PATCH_DIM1 + lit) + (LITERALS / 2)) / INT_SIZE;
            int neg_chunk_pos = ((DIM1 - PATCH_DIM1 + lit) + (LITERALS / 2)) % INT_SIZE;
            patch_output[neg_chunk_nr] &= ~(1u << neg_chunk_pos);
#endif
        }

        // Iterate over all pixels in the patch
        for (ull p_y = patch_coordinate_y; p_y < patch_coordinate_y + PATCH_DIM1; ++p_y) {
            for (ull p_x = patch_coordinate_x; p_x < patch_coordinate_x + PATCH_DIM0; ++p_x) {
                for (int z = 0; z < DIM2; ++z) {
                    unsigned long long dense_idx =
                        e * (ull)(DIM0 * DIM1 * DIM2) + p_y * (ull)(DIM0 * DIM2) + p_x * (ull)DIM2 + z;

                    if (X[dense_idx] > 0) {
                        int rel_y = p_y - patch_coordinate_y;
                        int rel_x = p_x - patch_coordinate_x;
#if ENCODE_LOC
                        int patch_pos =
                            (DIM1 - PATCH_DIM1) + (DIM0 - PATCH_DIM0) + rel_y * PATCH_DIM0 * DIM2 + rel_x * DIM2 + z;
#else
                        int patch_pos = rel_y * PATCH_DIM0 * DIM2 + rel_x * DIM2 + z;
#endif
                        int chunk_nr = patch_pos / INT_SIZE;
                        int chunk_pos = patch_pos % INT_SIZE;
                        patch_output[chunk_nr] |= (1u << chunk_pos);
#if APPEND_NEGATED
                        int neg_chunk_nr = (patch_pos + (LITERALS / 2)) / INT_SIZE;
                        int neg_chunk_pos = (patch_pos + (LITERALS / 2)) % INT_SIZE;
                        patch_output[neg_chunk_nr] &= ~(1u << neg_chunk_pos);
#endif
                    }
                }
            }
        }
    }
}

__global__ void pack_clauses(const unsigned int *global_ta_states, unsigned int *packed_clauses, int *num_includes) {
    /*
     * Pack the TA states into chunks of 32 bits. Each chunk represents a set of literals.
     * The number of included literals is also calculated here.
     */
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int clause = index; clause < CLAUSES; clause += stride) {
        const unsigned int *ta_state = &global_ta_states[clause * LITERALS];
        unsigned int *packed_clause = &packed_clauses[clause * NUM_LITERAL_CHUNKS];
        int total_count = 0;

#pragma unroll 4
        for (int chunk = 0; chunk < NUM_LITERAL_CHUNKS; ++chunk) {
            unsigned int packed_value = 0;
            int start_lit = chunk * INT_SIZE;
            int end_lit = min(start_lit + INT_SIZE, LITERALS);

            int vectorized_end = start_lit + ((end_lit - start_lit) & ~3);  // Ensure vectorized end is a multiple of 4
            for (int li = start_lit; li < vectorized_end; li += 4) {
                uint4 ta_vec = *((uint4 *)&ta_state[li]);
                if (ta_vec.x > HALF_STATE) {
                    packed_value |= (1u << (li % INT_SIZE));
                    total_count++;
                }
                if (ta_vec.y > HALF_STATE) {
                    packed_value |= (1u << ((li + 1) % INT_SIZE));
                    total_count++;
                }
                if (ta_vec.z > HALF_STATE) {
                    packed_value |= (1u << ((li + 2) % INT_SIZE));
                    total_count++;
                }
                if (ta_vec.w > HALF_STATE) {
                    packed_value |= (1u << ((li + 3) % INT_SIZE));
                    total_count++;
                }
            }
            for (int li = vectorized_end; li < end_lit; ++li) {
                if (ta_state[li] > HALF_STATE) {
                    packed_value |= (1u << (li % INT_SIZE));
                    total_count++;
                }
            }
            packed_clause[chunk] = packed_value;
        }
        num_includes[clause] = total_count;
    }
}

__device__ inline int clause_match_vec(const unsigned int *ta_state, const unsigned int *X) {
    int match = 1;
    const uint4 *ta_state_vec = reinterpret_cast<const uint4 *>(ta_state);
    const uint4 *X_vec = reinterpret_cast<const uint4 *>(X);

    int num_vec_chunks = (NUM_LITERAL_CHUNKS - 1) / 4;

#pragma unroll 4
    for (int vec_chunk = 0; vec_chunk < num_vec_chunks; vec_chunk++) {
        uint4 ta_vec = ta_state_vec[vec_chunk];
        uint4 x_vec = X_vec[vec_chunk];

        // Element-wise AND operations and comparisons
        match &= ((ta_vec.x & x_vec.x) == ta_vec.x) & ((ta_vec.y & x_vec.y) == ta_vec.y) &
                 ((ta_vec.z & x_vec.z) == ta_vec.z) & ((ta_vec.w & x_vec.w) == ta_vec.w);
    }

    // Handle remaining chunks that don't fit in uint4
    int remaining_start = num_vec_chunks * 4;

#pragma unroll
    for (int chunk = remaining_start; chunk < NUM_LITERAL_CHUNKS - 1; ++chunk) {
        match &= ((ta_state[chunk] & X[chunk]) == ta_state[chunk]);
    }

    match &= ((ta_state[NUM_LITERAL_CHUNKS - 1] & (X[NUM_LITERAL_CHUNKS - 1] & FILTER)) ==
              (ta_state[NUM_LITERAL_CHUNKS - 1] & FILTER));

    return match;
}

__device__ inline int clause_match_scalar(const unsigned int *ta_state, const unsigned int *X) {
#pragma unroll 4
    for (int chunk = 0; chunk < NUM_LITERAL_CHUNKS - 1; ++chunk)
        if ((ta_state[chunk] & X[chunk]) != ta_state[chunk]) return 0;

    if ((ta_state[NUM_LITERAL_CHUNKS - 1] & (X[NUM_LITERAL_CHUNKS - 1] & FILTER)) !=
        (ta_state[NUM_LITERAL_CHUNKS - 1] & FILTER))
        return 0;

    return 1;
}

__device__ inline int clause_match(const unsigned int *ta_state, const unsigned int *X) {
#if NUM_LITERAL_CHUNKS >= 4
    bool is_aligned = (((std::uintptr_t)ta_state % 16) == 0) && (((std::uintptr_t)X % 16) == 0);
    if (is_aligned)
        return clause_match_vec(ta_state, X);
    else
        return clause_match_scalar(ta_state, X);
#else
    return clause_match_scalar(ta_state, X);
#endif
}

__global__ void clause_eval(hiprandState *rng, const unsigned int *packed_ta_states, const float *clause_weights,
                            const unsigned int *X_batch, int *selected_patch_ids, float *class_sums, const int e) {
    /*
     * Calculate clause activations and select a patch for each active clause. If a clause is active, the
     * selected_patch_ids will be int between 0 and PATCHES - 1, else it will be -1.
     */
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState localRNG = rng[index];

    for (int clause = index; clause < CLAUSES; clause += stride) {
        int active_patches[PATCHES];
        int active_count = 0;

        for (int patch_id = 0; patch_id < PATCHES; ++patch_id) {
            int patch_matched =
                clause_match(&packed_ta_states[clause * NUM_LITERAL_CHUNKS],
                             &X_batch[(ull)e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * NUM_LITERAL_CHUNKS]);
            if (patch_matched) {
                active_patches[active_count] = patch_id;
                active_count++;
            }
        }
        if (active_count > 0) {
            int random_index = hiprand(&localRNG) % active_count;
            selected_patch_ids[clause] = active_patches[random_index];
            for (int class_id = 0; class_id < CLASSES; ++class_id) {
                atomicAdd(&class_sums[0 * CLASSES + class_id], clause_weights[clause * CLASSES + class_id]);
            }
        } else {
            selected_patch_ids[clause] = -1;
        }
    }
    rng[index] = localRNG;
}

__global__ void calc_class_sums_infer_batch(const unsigned int *packed_ta_states, const float *clause_weights,
                                            const int *num_includes, const unsigned int *X_batch, const int N,
                                            float *class_sums_batch) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (ull e_clause = index; e_clause < (ull)N * (ull)CLAUSES; e_clause += stride) {
        ull e = e_clause / CLAUSES;
        ull clause = e_clause % CLAUSES;
        if (num_includes[clause] == 0) continue;  // Skip empty clauses
        int clause_output = 0;
        for (int patch_id = 0; patch_id < PATCHES; ++patch_id) {
            if (clause_match(&packed_ta_states[clause * NUM_LITERAL_CHUNKS],
                             &X_batch[e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * NUM_LITERAL_CHUNKS])) {
                clause_output = 1;
                break;
            }
        }
        if (clause_output) {
            for (int class_id = 0; class_id < CLASSES; ++class_id) {
                atomicAdd(&class_sums_batch[e * CLASSES + class_id], clause_weights[clause * CLASSES + class_id]);
            }
        }
    }
}

__device__ inline float clip_cs(float cs) { return (cs > THRESH) ? THRESH : ((cs < -THRESH) ? -THRESH : cs); }

__device__ inline void type1a_fb(hiprandState *rng, unsigned int *ta_state, const unsigned int *patch) {
    /*
     * Type Ia feedback - Vectorized version:
     * 1. Increment states for literal present in the patch.
     * 2. Decrement states for literal not present in the patch with probability 1/S.
     * 3. Increase clause weight.
     */

#pragma unroll 4
    for (int li = 0; li < VECTORIZED_LIMIT; li += 4) {
        uint4 ta_vec = *((uint4 *)&ta_state[li]);
        uint4 patch_vec = {
            (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u,
            (patch[(li + 1) / INT_SIZE] >> ((li + 1) % INT_SIZE)) & 1u,
            (patch[(li + 2) / INT_SIZE] >> ((li + 2) % INT_SIZE)) & 1u,
            (patch[(li + 3) / INT_SIZE] >> ((li + 3) % INT_SIZE)) & 1u,
        };

        ta_vec.x += (patch_vec.x == 1 && ta_vec.x < MAX_TA_STATE);
        ta_vec.y += (patch_vec.y == 1 && ta_vec.y < MAX_TA_STATE);
        ta_vec.z += (patch_vec.z == 1 && ta_vec.z < MAX_TA_STATE);
        ta_vec.w += (patch_vec.w == 1 && ta_vec.w < MAX_TA_STATE);

        ta_vec.x -= (patch_vec.x == 0 && ta_vec.x > 0 && hiprand_uniform(rng) <= S_INV);
        ta_vec.y -= (patch_vec.y == 0 && ta_vec.y > 0 && hiprand_uniform(rng) <= S_INV);
        ta_vec.z -= (patch_vec.z == 0 && ta_vec.z > 0 && hiprand_uniform(rng) <= S_INV);
        ta_vec.w -= (patch_vec.w == 0 && ta_vec.w > 0 && hiprand_uniform(rng) <= S_INV);

        // Write back the vectorized results
        *((uint4 *)&ta_state[li]) = ta_vec;
    }

// Handle remaining literals (when LITERALS % 4 != 0)
#pragma unroll 4
    for (int li = VECTORIZED_LIMIT; li < LITERALS; ++li) {
        unsigned int patch_bit = (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u;
        if (patch_bit == 1 && ta_state[li] < MAX_TA_STATE) {
            ta_state[li] += 1;
        } else if (patch_bit == 0 && ta_state[li] > 0 && hiprand_uniform(rng) < S_INV) {
            ta_state[li] -= 1;
        }
    }
}

__device__ inline void type1b_fb(hiprandState *rng, unsigned int *ta_state) {
    /*
     * Type Ib feedback - Vectorized version:
     * 1. Decrement states for all literals with probability 1/S.
     */

#pragma unroll 4
    for (int li = 0; li < VECTORIZED_LIMIT; li += 4) {
        uint4 ta_vec = *((uint4 *)&ta_state[li]);

        ta_vec.x -= (ta_vec.x > 0 && hiprand_uniform(rng) <= S_INV);
        ta_vec.y -= (ta_vec.y > 0 && hiprand_uniform(rng) <= S_INV);
        ta_vec.z -= (ta_vec.z > 0 && hiprand_uniform(rng) <= S_INV);
        ta_vec.w -= (ta_vec.w > 0 && hiprand_uniform(rng) <= S_INV);

        *((uint4 *)&ta_state[li]) = ta_vec;
    }

#pragma unroll 4
    for (int li = VECTORIZED_LIMIT; li < LITERALS; ++li) {
        if (ta_state[li] > 0 && hiprand_uniform(rng) < S_INV) {
            ta_state[li] -= 1;
        }
    }
}

__device__ inline void type2_fb(unsigned int *ta_state, const unsigned int *patch) {
    /*
     * Type II feedback - Vectorized version with macro constants:
     * 1. Increment states for literals not present in patch.
     * 2. Decrement clause weight.
     */

// Use predefined macro instead of runtime computation
#pragma unroll 4
    for (int li = 0; li < VECTORIZED_LIMIT; li += 4) {
        uint4 ta_vec = *((uint4 *)&ta_state[li]);
        uint4 patch_vec = {
            (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u,
            (patch[(li + 1) / INT_SIZE] >> ((li + 1) % INT_SIZE)) & 1u,
            (patch[(li + 2) / INT_SIZE] >> ((li + 2) % INT_SIZE)) & 1u,
            (patch[(li + 3) / INT_SIZE] >> ((li + 3) % INT_SIZE)) & 1u,
        };

        // Increment ta_state elements where patch is 0
        ta_vec.x += (patch_vec.x == 0);
        ta_vec.y += (patch_vec.y == 0);
        ta_vec.z += (patch_vec.z == 0);
        ta_vec.w += (patch_vec.w == 0);

        *((uint4 *)&ta_state[li]) = ta_vec;
    }

// Handle remaining literals using macro constant
#pragma unroll 4
    for (int li = VECTORIZED_LIMIT; li < LITERALS; ++li) {
        unsigned int patch_bit = (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u;
        if (patch_bit == 0) {
            ta_state[li] += 1;
        }
    }
}

__global__ void clause_update(hiprandState *rng, unsigned int *global_ta_states, float *clause_weights,
                              const float *class_sums, const int *selected_patch_ids, const int *num_includes,
                              const unsigned int *X_batch, const int *Y_batch, const int e) {
    /*
     * Update the clauses based on the class sum and Y.
     *
     * Params:
     * - rng: Random number generator.
     * - global_ta_states: TA states. Shape: CLAUSES * LITERALS.
     * - clause_weights: Weights of the clauses. Shape: CLAUSES * CLASSES.
     * - class_sums: Shape: CLASSES.
     * - selected_patch_ids: Selected patch ids for each clause. Shape: CLAUSES.
     * - num_includes: Number of included literals for each clause. Shape: CLAUSES.
     * - X: Packed Input data. Shape: PATCHES * NUM_LITERAL_CHUNKS.
     * - Y: Labels for the sample. This is encoded, i.e., y == 0 -> -T and y == 1 -> T. Shape: CLASSES.
     */

    __shared__ float prob[CLASSES];
    __shared__ int tar[CLASSES];
    for (int class_id = threadIdx.x; class_id < CLASSES; class_id += blockDim.x) {
        float clipped = clip_cs(class_sums[class_id]);
        const int y = Y_batch[e * CLASSES + class_id];
        prob[class_id] = abs((float)y - clipped) / (2.0 * THRESH);
        tar[class_id] = 1 - 2 * (clipped > y);
    }
    __syncthreads();

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState localRNG = rng[index];

    for (int clause = index; clause < CLAUSES; clause += stride) {
        unsigned int *ta_state = &global_ta_states[clause * LITERALS];
        int local_clause_output = selected_patch_ids[clause] > -1 ? 1 : 0;
        const unsigned int *X = &X_batch[(ull)e * (ull)(PATCHES * NUM_LITERAL_CHUNKS)];
        const unsigned int *patch =
            selected_patch_ids[clause] > -1 ? &X[selected_patch_ids[clause] * NUM_LITERAL_CHUNKS] : nullptr;

        for (ull class_id = 0; class_id < CLASSES; ++class_id) {
            if (tar[class_id] == -1 && hiprand_uniform(&localRNG) > Q_PROB) {
                continue;  // Skip the class.
            }

            float *local_weight = &clause_weights[clause * CLASSES + class_id];
            int sign = (*local_weight >= 0) - (*local_weight < 0);
            bool should_upate = (hiprand_uniform(&localRNG) <= prob[class_id]);
            bool type1a = ((tar[class_id] * sign) > 0 && local_clause_output);
            bool type1b = ((tar[class_id] * sign) > 0 && !local_clause_output);
            bool type2 = ((tar[class_id] * sign) < 0 && local_clause_output);

            if (should_upate) {  // CLause update with prob update_p else skip
                if (type1a) {
                    (*local_weight) += sign * 1.0f;
#if (MAX_INCLUDED_LITERALS < LITERALS)
                    if (num_includes[clause] < MAX_INCLUDED_LITERALS) type1a_fb(&localRNG, ta_state, patch);
#else
                    type1a_fb(&localRNG, ta_state, patch);
#endif
                } else if (type1b) {
                    type1b_fb(&localRNG, ta_state);
                } else if (type2) {
                    (*local_weight) -= sign * 1.0f;
#if NEGATIVE_CLAUSES == 0
                    if (*local_weight < 1) *local_weight = 1;
#endif
                    type2_fb(ta_state, patch);
                }
            }
        }
    }
    rng[index] = localRNG;
}
}
