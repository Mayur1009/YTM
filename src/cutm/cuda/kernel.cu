#include "hip/hip_runtime.h"
// Ignore this block, it is used to only for neovim clangd lsp.
#ifdef IS_NEOVIM_CLANGD_ENV
    #define CLAUSES 100
    #define THRESH 500
    #define S 10
    #define Q 1
    #define DIM0 28
    #define DIM1 28
    #define DIM2 1
    #define PATCH_DIM0 10
    #define PATCH_DIM1 10
    #define PATCHES 361
    #define LITERALS 272
    #define MAX_INCLUDED_LITERALS 272
    #define APPEND_NEGATED 1
    #define INIT_NEG_WEIGHTS 1
    #define NEGATIVE_CLAUSES 1
    #define CLASSES 10
    #define MAX_TA_STATE 255
    #define ENCODE_LOC 1
    #define COALESCED 1
    #define CLAUSE_BANKS 1
__constant__ const double H[CLASSES] = {1};
    #define BIAS 0
#endif

#include <hiprand/hiprand_kernel.h>

#define CLAUSES_PER_BANK (CLAUSES / CLAUSE_BANKS)
#define VECTORIZED_LIMIT (LITERALS & ~3)
#define S_INV (1.0f / S)
#define Q_PROB (1.0f * Q / max(1, CLASSES - 1))
#define HALF_STATE (MAX_TA_STATE / 2)
#define INT_SIZE 32
#define NUM_LITERAL_CHUNKS (((LITERALS - 1) / INT_SIZE) + 1)
#if ((LITERALS % INT_SIZE) != 0)
    #define FILTER (~(0xFFFFFFFF << (LITERALS % INT_SIZE)))
#else
    #define FILTER 0xFFFFFFFF
#endif

typedef unsigned long long ull;

extern "C" {
    /***********INITIALIZATION***********/
    __global__ void init_weights(hiprandState *rng, float *clause_weights) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        hiprandState localState = rng[index];

        for (int clause = index; clause < CLAUSES; clause += stride) {
            for (int class_id = 0; class_id < CLASSES; ++class_id) {
#if COALESCED  // Coalesced -- all clauses have weight for all classes.
                clause_weights[clause * CLASSES + class_id] = 1.0f;
    #if INIT_NEG_WEIGHTS
                clause_weights[clause * CLASSES + class_id] = (1.0f - 2.0f * (float)(hiprand(&localState) % 2));
    #endif
#else
                if (class_id == clause / CLAUSES_PER_BANK) {  // Clause belongs to this class.
                    clause_weights[clause * CLASSES + class_id] = 1.0f;
    #if INIT_NEG_WEIGHTS  // Initialize negative polarity in second half of the clause bank.
                    if ((clause % CLAUSES_PER_BANK) >= (CLAUSES_PER_BANK / 2))
                        clause_weights[clause * CLASSES + class_id] = -1.0f;
    #endif
                } else {
                    clause_weights[clause * CLASSES + class_id] = 0.0f;
                }
#endif
            }
        }

        rng[index] = localState;
    }

    /***********INPUT ENCODING***********/
    __global__ void encode_batch(const unsigned int *X, unsigned int *encoded_X, const int N) {
        // X -> (N * DIM0 * DIM1 * DIM2)
        // encoded_X -> (N * PATCHES * NUM_LITERAL_CHUNKS)
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (ull e_patch = index; e_patch < (ull)(PATCHES * N); e_patch += stride) {
            ull e = e_patch / PATCHES;
            ull patch_id = e_patch % PATCHES;

            // Calculate the starting point of the patch in the original image
            int patch_coordinate_y = patch_id / (DIM0 - PATCH_DIM0 + 1);
            int patch_coordinate_x = patch_id % (DIM0 - PATCH_DIM0 + 1);

            ull encX_offset = e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * (ull)NUM_LITERAL_CHUNKS;
            unsigned int *patch_output = &encoded_X[encX_offset];

#if APPEND_NEGATED
            for (int literal = LITERALS / 2; literal < LITERALS; ++literal) {
                int chunk_nr = literal / INT_SIZE;
                int chunk_pos = literal % INT_SIZE;
                patch_output[chunk_nr] |= (1u << chunk_pos);
            }
#endif

            for (int lit = 0; lit < patch_coordinate_y; ++lit) {
                int chunk_nr = lit / INT_SIZE;
                int chunk_pos = lit % INT_SIZE;
                patch_output[chunk_nr] |= (1u << chunk_pos);
#if APPEND_NEGATED
                int neg_chunk_nr = (lit + (LITERALS / 2)) / INT_SIZE;
                int neg_chunk_pos = (lit + (LITERALS / 2)) % INT_SIZE;
                patch_output[neg_chunk_nr] &= ~(1u << neg_chunk_pos);
#endif
            }

            for (int lit = 0; lit < patch_coordinate_x; ++lit) {
                int chunk_nr = (DIM1 - PATCH_DIM1 + lit) / INT_SIZE;
                int chunk_pos = (DIM1 - PATCH_DIM1 + lit) % INT_SIZE;
                patch_output[chunk_nr] |= (1u << chunk_pos);
#if APPEND_NEGATED
                int neg_chunk_nr = ((DIM1 - PATCH_DIM1 + lit) + (LITERALS / 2)) / INT_SIZE;
                int neg_chunk_pos = ((DIM1 - PATCH_DIM1 + lit) + (LITERALS / 2)) % INT_SIZE;
                patch_output[neg_chunk_nr] &= ~(1u << neg_chunk_pos);
#endif
            }

            // Iterate over all pixels in the patch
            for (ull p_y = patch_coordinate_y; p_y < patch_coordinate_y + PATCH_DIM1; ++p_y) {
                for (ull p_x = patch_coordinate_x; p_x < patch_coordinate_x + PATCH_DIM0; ++p_x) {
                    for (int z = 0; z < DIM2; ++z) {
                        unsigned long long dense_idx =
                            e * (ull)(DIM0 * DIM1 * DIM2) + p_y * (ull)(DIM0 * DIM2) + p_x * (ull)DIM2 + z;

                        if (X[dense_idx] > 0) {
                            int rel_y = p_y - patch_coordinate_y;
                            int rel_x = p_x - patch_coordinate_x;
#if ENCODE_LOC
                            int patch_pos = (DIM1 - PATCH_DIM1) + (DIM0 - PATCH_DIM0) + rel_y * PATCH_DIM0 * DIM2 +
                                            rel_x * DIM2 + z;
#else
                            int patch_pos = rel_y * PATCH_DIM0 * DIM2 + rel_x * DIM2 + z;
#endif
                            int chunk_nr = patch_pos / INT_SIZE;
                            int chunk_pos = patch_pos % INT_SIZE;
                            patch_output[chunk_nr] |= (1u << chunk_pos);
#if APPEND_NEGATED
                            int neg_chunk_nr = (patch_pos + (LITERALS / 2)) / INT_SIZE;
                            int neg_chunk_pos = (patch_pos + (LITERALS / 2)) % INT_SIZE;
                            patch_output[neg_chunk_nr] &= ~(1u << neg_chunk_pos);
#endif
                        }
                    }
                }
            }
        }
    }

    /***********CLAUSE PACKING***********/
    __global__ void pack_clauses(const unsigned int *global_ta_states, unsigned int *packed_clauses,
                                 int *num_includes) {
        /*
         * Pack the TA states into chunks of 32 bits. Each chunk represents a set of literals.
         * The number of included literals is also calculated here.
         */
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int clause = index; clause < CLAUSES; clause += stride) {
            const unsigned int *ta_state = &global_ta_states[clause * LITERALS];
            unsigned int *packed_clause = &packed_clauses[clause * NUM_LITERAL_CHUNKS];
            int total_count = 0;

            for (int chunk = 0; chunk < NUM_LITERAL_CHUNKS; ++chunk) {
                unsigned int packed_value = 0;
                int start_lit = chunk * INT_SIZE;
                int end_lit = min(start_lit + INT_SIZE, LITERALS);

                int vectorized_end =
                    start_lit + ((end_lit - start_lit) & ~3);  // Ensure vectorized end is a multiple of 4
                for (int li = start_lit; li < vectorized_end; li += 4) {
                    uint4 ta_vec = *((uint4 *)&ta_state[li]);
                    if (ta_vec.x > HALF_STATE) {
                        packed_value |= (1u << (li % INT_SIZE));
                        total_count++;
                    }
                    if (ta_vec.y > HALF_STATE) {
                        packed_value |= (1u << ((li + 1) % INT_SIZE));
                        total_count++;
                    }
                    if (ta_vec.z > HALF_STATE) {
                        packed_value |= (1u << ((li + 2) % INT_SIZE));
                        total_count++;
                    }
                    if (ta_vec.w > HALF_STATE) {
                        packed_value |= (1u << ((li + 3) % INT_SIZE));
                        total_count++;
                    }
                }
                for (int li = vectorized_end; li < end_lit; ++li) {
                    if (ta_state[li] > HALF_STATE) {
                        packed_value |= (1u << (li % INT_SIZE));
                        total_count++;
                    }
                }
                packed_clause[chunk] = packed_value;
            }
            num_includes[clause] = total_count;
        }
    }

    /***********CLAUSE EVALUATION***********/
    __device__ inline int clause_match(const unsigned int *ta_state, const unsigned int *X) {
        for (int chunk = 0; chunk < NUM_LITERAL_CHUNKS - 1; ++chunk)
            if ((ta_state[chunk] & X[chunk]) != ta_state[chunk]) return 0;

        if ((ta_state[NUM_LITERAL_CHUNKS - 1] & (X[NUM_LITERAL_CHUNKS - 1] & FILTER)) !=
            (ta_state[NUM_LITERAL_CHUNKS - 1] & FILTER))
            return 0;

        return 1;
    }

    /***********CLAUSE EVALUATION---SLOWER***********/
    __global__ void clause_eval(hiprandState *rng, const unsigned int *packed_ta_states, const float *clause_weights,
                                int *patch_weights, const unsigned int *X_batch, int *selected_patch_ids,
                                float *class_sums, const int e) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        hiprandState localRNG = rng[index];

        for (int clause = index; clause < CLAUSES; clause += stride) {
            int active_patches[PATCHES];
            int active_count = 0;

            for (int patch_id = 0; patch_id < PATCHES; ++patch_id) {
                int patch_matched = clause_match(
                    &packed_ta_states[clause * NUM_LITERAL_CHUNKS],
                    &X_batch[(ull)e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * NUM_LITERAL_CHUNKS]);
                if (patch_matched) {
                    active_patches[active_count] = patch_id;
                    active_count++;
                }
            }
            if (active_count > 0) {
                int random_index = hiprand(&localRNG) % active_count;
                selected_patch_ids[clause] = active_patches[random_index];
                patch_weights[clause * PATCHES + active_patches[random_index]] = 1;
                for (int class_id = 0; class_id < CLASSES; ++class_id) {
                    atomicAdd(&class_sums[0 * CLASSES + class_id], clause_weights[clause * CLASSES + class_id]);
                }
            } else {
                selected_patch_ids[clause] = -1;
            }
        }
        rng[index] = localRNG;
    }

    /***********FAST EVALUATION KERNELS***********/
    __global__ void fast_eval(const unsigned int *packed_ta_states, const int *num_includes,
                              const unsigned int *clause_drop_mask, const unsigned int *X_batch,
                              unsigned int *clause_outputs, const int e) {
        // clause_outputs => (N * CLAUSES * PATCHES)
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (ull clause_patch = index; clause_patch < (ull)CLAUSES * (ull)PATCHES; clause_patch += stride) {
            unsigned int *clause_output = &clause_outputs[clause_patch];

            ull clause = clause_patch / PATCHES;
            ull patch_id = clause_patch % PATCHES;

            // Skip dropped clauses
            if (clause_drop_mask[clause] == 1) {
                *clause_output = 0;
                continue;
            }

            if (num_includes[clause] == 0) {
                *clause_output = 1;
                continue;
            }

            *clause_output =
                clause_match(&packed_ta_states[clause * NUM_LITERAL_CHUNKS],
                             &X_batch[(ull)e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * NUM_LITERAL_CHUNKS]);
        }
    }

    /***********SELECT ACTIVE CLAUSES AND CALCULATE CLASS SUMS***********/
    __global__ void select_active(hiprandState *rng, const float *clause_weights, const unsigned int *clause_outputs,
                                  int *patch_weights, int *selected_patch_ids, float *class_sums) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        hiprandState localRNG = rng[index];

        for (int clause = index; clause < CLAUSES; clause += stride) {
            int count = 0;
            int selected_id = -1;
            for (int patch_id = 0; patch_id < PATCHES; ++patch_id) {
                if (clause_outputs[clause * PATCHES + patch_id]) {
                    count++;
                    if (hiprand_uniform(&localRNG) < 1.0f / count) {
                        selected_id = patch_id;
                    }
                }
            }
            selected_patch_ids[clause] = selected_id;
            if (selected_id != -1) {
                patch_weights[clause * PATCHES + selected_id]++;
#if COALESCED == 0
                int class_id = (ull)clause / CLAUSES_PER_BANK;
#else
                for (int class_id = 0; class_id < CLASSES; ++class_id)
#endif
                atomicAdd(&class_sums[class_id],
                          clause_outputs[clause * PATCHES + selected_id] * clause_weights[clause * CLASSES + class_id]);
            }
        }
        rng[index] = localRNG;
    }

    /***********FAST CLASS SUMS CALCULATION FOR INFERENCE***********/
    __global__ void calc_class_sums_infer_batch(const unsigned int *packed_ta_states, const float *clause_weights,
                                                const int *num_includes, const unsigned int *X_batch, const int N,
                                                float *class_sums_batch) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (ull e_clause = index; e_clause < (ull)N * (ull)CLAUSES; e_clause += stride) {
            ull e = e_clause / CLAUSES;
            ull clause = e_clause % CLAUSES;
            if (num_includes[clause] == 0) continue;  // Skip empty clauses
            int clause_output = 0;
            for (int patch_id = 0; patch_id < PATCHES; ++patch_id) {
                if (clause_match(&packed_ta_states[clause * NUM_LITERAL_CHUNKS],
                                 &X_batch[e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * NUM_LITERAL_CHUNKS])) {
                    clause_output = 1;
                    break;
                }
            }
            if (clause_output) {
                for (int class_id = 0; class_id < CLASSES; ++class_id) {
                    atomicAdd(&class_sums_batch[e * CLASSES + class_id], clause_weights[clause * CLASSES + class_id]);
                }
            }
        }
    }

    /***********TRNAFORM KERNELS***********/
    __global__ void transform(const unsigned int *packed_ta_states, const int *num_includes,
                              const unsigned int *X_batch, const int N, unsigned int *clause_outputs) {
        // clause_outputs => (N * CLAUSES)
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (ull e_clause = index; e_clause < (ull)N * (ull)CLAUSES; e_clause += stride) {
            ull e = e_clause / CLAUSES;
            ull clause = e_clause % CLAUSES;
            if (num_includes[clause] == 0) {
                clause_outputs[e * CLAUSES + clause] = 1;
                continue;
            }
            int clause_output = 0;
            for (int patch_id = 0; patch_id < PATCHES; ++patch_id) {
                if (clause_match(&packed_ta_states[clause * NUM_LITERAL_CHUNKS],
                                 &X_batch[e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * NUM_LITERAL_CHUNKS])) {
                    clause_output = 1;
                    break;
                }
            }
            clause_outputs[e * CLAUSES + clause] = clause_output;
        }
    }

    __global__ void transform_patchwise(const unsigned int *packed_ta_states, const int *num_includes,
                                        const unsigned int *X_batch, const int N, unsigned int *clause_outputs) {
        // clause_outputs => (N * CLAUSES * PATCHES)
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (ull e_clause_patch = index; e_clause_patch < (ull)N * (ull)CLAUSES * (ull)PATCHES;
             e_clause_patch += stride) {
            unsigned int *clause_output = &clause_outputs[e_clause_patch];

            ull e_clause = e_clause_patch / PATCHES;
            ull patch_id = e_clause_patch % PATCHES;

            ull e = e_clause / CLAUSES;
            ull clause = e_clause % CLAUSES;

            if (num_includes[clause] == 0) {
                *clause_output = 1;
                continue;
            }

            *clause_output =
                clause_match(&packed_ta_states[clause * NUM_LITERAL_CHUNKS],
                             &X_batch[e * (ull)(PATCHES * NUM_LITERAL_CHUNKS) + patch_id * NUM_LITERAL_CHUNKS]);
        }
    }

    /***********CLAUSE UPDATE KERNELS***********/
    __device__ inline float clip_cs(float cs) { return (cs > THRESH) ? THRESH : ((cs < -THRESH) ? -THRESH : cs); }

    __device__ inline void type1a_fb_scalar(hiprandState *rng, unsigned int *ta_state, const unsigned int *patch) {
        for (int li = 0; li < LITERALS; ++li) {
            unsigned int patch_bit = (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u;
            if (patch_bit == 1 && ta_state[li] < MAX_TA_STATE) {
                ta_state[li] += 1;
            } else if (patch_bit == 0 && ta_state[li] > 0 && hiprand_uniform(rng) <= S_INV) {
                ta_state[li] -= 1;
            }
        }
    }

    __device__ inline void type1a_fb(hiprandState *rng, unsigned int *ta_state, const unsigned int *patch) {
        for (int li = 0; li < VECTORIZED_LIMIT; li += 4) {
            uint4 ta_vec = *((uint4 *)&ta_state[li]);
            uint4 patch_vec = {
                (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u,
                (patch[(li + 1) / INT_SIZE] >> ((li + 1) % INT_SIZE)) & 1u,
                (patch[(li + 2) / INT_SIZE] >> ((li + 2) % INT_SIZE)) & 1u,
                (patch[(li + 3) / INT_SIZE] >> ((li + 3) % INT_SIZE)) & 1u,
            };

            ta_vec.x += (patch_vec.x == 1 && ta_vec.x < MAX_TA_STATE);
            ta_vec.y += (patch_vec.y == 1 && ta_vec.y < MAX_TA_STATE);
            ta_vec.z += (patch_vec.z == 1 && ta_vec.z < MAX_TA_STATE);
            ta_vec.w += (patch_vec.w == 1 && ta_vec.w < MAX_TA_STATE);

            ta_vec.x -= (patch_vec.x == 0 && ta_vec.x > 0 && hiprand_uniform(rng) <= S_INV);
            ta_vec.y -= (patch_vec.y == 0 && ta_vec.y > 0 && hiprand_uniform(rng) <= S_INV);
            ta_vec.z -= (patch_vec.z == 0 && ta_vec.z > 0 && hiprand_uniform(rng) <= S_INV);
            ta_vec.w -= (patch_vec.w == 0 && ta_vec.w > 0 && hiprand_uniform(rng) <= S_INV);

            // Write back the vectorized results
            *((uint4 *)&ta_state[li]) = ta_vec;
        }

        // Handle remaining literals (when LITERALS % 4 != 0)
        for (int li = VECTORIZED_LIMIT; li < LITERALS; ++li) {
            unsigned int patch_bit = (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u;
            if (patch_bit == 1 && ta_state[li] < MAX_TA_STATE) {
                ta_state[li] += 1;
            } else if (patch_bit == 0 && ta_state[li] > 0 && hiprand_uniform(rng) <= S_INV) {
                ta_state[li] -= 1;
            }
        }
    }

    __device__ inline void type1b_fb_scalar(hiprandState *rng, unsigned int *ta_state) {
        for (int li = 0; li < LITERALS; ++li) {
            if (ta_state[li] > 0 && hiprand_uniform(rng) <= S_INV) {
                ta_state[li] -= 1;
            }
        }
    }

    __device__ inline void type1b_fb(hiprandState *rng, unsigned int *ta_state) {
        for (int li = 0; li < VECTORIZED_LIMIT; li += 4) {
            uint4 ta_vec = *((uint4 *)&ta_state[li]);

            ta_vec.x -= (ta_vec.x > 0 && hiprand_uniform(rng) <= S_INV);
            ta_vec.y -= (ta_vec.y > 0 && hiprand_uniform(rng) <= S_INV);
            ta_vec.z -= (ta_vec.z > 0 && hiprand_uniform(rng) <= S_INV);
            ta_vec.w -= (ta_vec.w > 0 && hiprand_uniform(rng) <= S_INV);

            *((uint4 *)&ta_state[li]) = ta_vec;
        }

        for (int li = VECTORIZED_LIMIT; li < LITERALS; ++li) {
            if (ta_state[li] > 0 && hiprand_uniform(rng) <= S_INV) {
                ta_state[li] -= 1;
            }
        }
    }

    __device__ inline void type2_fb_scalar(unsigned int *ta_state, const unsigned int *patch) {
        for (int li = 0; li < LITERALS; ++li) {
            unsigned int patch_bit = (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u;
            if (patch_bit == 0 && ta_state[li] <= HALF_STATE) {
                ta_state[li] += 1;
            }
        }
    }

    __device__ inline void type2_fb(unsigned int *ta_state, const unsigned int *patch) {
        for (int li = 0; li < VECTORIZED_LIMIT; li += 4) {
            uint4 ta_vec = *((uint4 *)&ta_state[li]);
            uint4 patch_vec = {
                (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u,
                (patch[(li + 1) / INT_SIZE] >> ((li + 1) % INT_SIZE)) & 1u,
                (patch[(li + 2) / INT_SIZE] >> ((li + 2) % INT_SIZE)) & 1u,
                (patch[(li + 3) / INT_SIZE] >> ((li + 3) % INT_SIZE)) & 1u,
            };

            // Increment ta_state elements where patch is 0
            ta_vec.x += (patch_vec.x == 0);
            ta_vec.y += (patch_vec.y == 0);
            ta_vec.z += (patch_vec.z == 0);
            ta_vec.w += (patch_vec.w == 0);

            *((uint4 *)&ta_state[li]) = ta_vec;
        }

        for (int li = VECTORIZED_LIMIT; li < LITERALS; ++li) {
            unsigned int patch_bit = (patch[li / INT_SIZE] >> (li % INT_SIZE)) & 1u;
            if (patch_bit == 0) {
                ta_state[li] += 1;
            }
        }
    }

    __device__ inline double update_probability(double v, double y, double mod, double h) {
        double prob;
        // if (y > 0) {
        //     if (v <= y * (2 * h - 1))
        //         prob = (y - v) / (2 * y);
        //     else {
        //         double a = (1 - h);
        //         double b = (y - v) / (2 * y * a);
        //         prob = a * (1 - pow(1 - b, 1.0 / mod));
        //     }
        // } else {
        //     if (v > y * (2 * h - 1))
        //         prob = (y - v) / (2 * y);
        //     else {
        //         double a = (1 - h);
        //         double b = (y - v) / (2 * y * a);
        //         prob = a * (1 - pow(1 - b, 1.0 / mod));
        //     }
        // }
        prob = (y - v) / (2 * y);
        return prob;
    }

    __global__ void clause_update(hiprandState *rng, unsigned int *global_ta_states, float *clause_weights,
                                  float *bias_weights, const float *class_sums, const int *selected_patch_ids,
                                  const int *num_includes, const double *true_mod, const double *false_mod,
                                  const unsigned int *clause_drop_mask, const unsigned int *X_batch, const int *Y_batch,
                                  const int e, const int focusced_pos_sampling, const int focused_neg_sampling) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        hiprandState localRNG = rng[index];

        // Should this be separate kernel?
        double update_probs[CLASSES];
        double pos_target_sum = 0, neg_target_sum = 0;
        for (int class_id = 0; class_id < CLASSES; ++class_id) {
            float clipped_cs = clip_cs(class_sums[class_id]);
            int y = Y_batch[e * CLASSES + class_id];
            int local_target = 1 - 2 * (clipped_cs > y);
            update_probs[class_id] =
                update_probability((double)clipped_cs, (double)y,
                                   local_target == 1 ? true_mod[class_id] : false_mod[class_id], H[class_id]);
            local_target == 1 ? (pos_target_sum += update_probs[class_id]) : (neg_target_sum += update_probs[class_id]);
        }

        for (int clause = index; clause < CLAUSES; clause += stride) {
            // Skip dropped clauses
            if (clause_drop_mask[clause] == 1) continue;

            unsigned int *ta_state = &global_ta_states[clause * LITERALS];
            int local_clause_output = selected_patch_ids[clause] > -1 ? 1 : 0;
            const unsigned int *X = &X_batch[(ull)e * (ull)(PATCHES * NUM_LITERAL_CHUNKS)];
            const unsigned int *patch =
                selected_patch_ids[clause] > -1 ? &X[selected_patch_ids[clause] * NUM_LITERAL_CHUNKS] : nullptr;

#if COALESCED == 0
            ull class_id = (ull)clause / CLAUSES_PER_BANK;
            {
#else
            for (ull class_id = 0; class_id < CLASSES; ++class_id) {
#endif
                float clipped_cs = clip_cs(class_sums[class_id]);
                int y = Y_batch[e * CLASSES + class_id];
                int local_target = 1 - 2 * (clipped_cs > y);

                if (local_target == -1 && hiprand_uniform(&localRNG) > Q_PROB) continue;

                double update_prob = update_probs[class_id];
                // if (focusced_pos_sampling && local_target == 1) update_prob = update_prob / pos_target_sum;
                // if (focused_neg_sampling && local_target == -1) update_prob = update_prob / neg_target_sum;

                float *local_weight = &clause_weights[clause * CLASSES + class_id];
                int sign = (*local_weight >= 0) - (*local_weight < 0);

                bool should_update = (hiprand_uniform(&localRNG) <= update_prob);
                bool type1a =
                    ((local_target * sign) > 0 && local_clause_output && num_includes[clause] <= MAX_INCLUDED_LITERALS);
                bool type1b = ((local_target * sign) > 0 &&
                               !(local_clause_output && num_includes[clause] <= MAX_INCLUDED_LITERALS));
                bool type2 = ((local_target * sign) < 0 && local_clause_output);

                if (should_update) {  // CLause update with prob update_p else skip
                    if (type1a) {
                        (*local_weight) += sign * 1.0f;
#if BIAS
                        bias_weights[class_id] += sign * 1.0f;
#endif
                        type1a_fb(&localRNG, ta_state, patch);
                    } else if (type1b) {
                        type1b_fb(&localRNG, ta_state);
                    } else if (type2) {
                        (*local_weight) -= sign * 1.0f;
#if BIAS
                        bias_weights[class_id] -= sign * 1.0f;
#endif
#if NEGATIVE_CLAUSES == 0
                        if (*local_weight < 1) *local_weight = 1;
                        if (bias_weights[class_id] < 0) bias_weights[class_id] = 0;
#endif
                        type2_fb(ta_state, patch);
                    }
                }
            }
        }
        rng[index] = localRNG;
    }
}
